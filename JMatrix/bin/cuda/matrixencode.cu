//CUDA


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


int main(int argc, char* argv[]){
	printf("Hello world\n");
	// Read matrices metadata from stdin

	// format:
	//	nnn x mmm - height x width
	
	int n, m;

	// Allocate memory (both on host & device)
	// 	Allocate host memory
	
	// Array of pointers;
	double *host_matrix[n];
	for(int i = 0; i < n; i++){
		host_matrix[i] = (double *)malloc(m*sizeof(double));
	}
	// useful thing: a[i][j] = *(*(a+i)+j)
	
	// Populate host memory structure with data - streamed in via stdin

	
	//	Allocate device memory
	
	double* dev_matrix;
	// serialise data for easiness
	hipMalloc((void **)&dev_matrix, sizeof(double)*m*n);
		
	// read the matrix data from stdin
	// perform accelerated matrix calculation
	// write output (& metadata) to stdout
	return 0;
}
